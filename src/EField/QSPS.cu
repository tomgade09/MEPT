#include "hip/hip_runtime.h"
#include "EField/QSPS.h"

#include <filesystem>

#include ""
#include "utils/serializationHelpers.h"
#include "ErrorHandling/cudaErrorCheck.h"
#include "ErrorHandling/cudaDeviceMacros.h"

using std::cerr;
using std::to_string;
using std::invalid_argument;
using namespace utils::fileIO::serialize;

namespace QSPS_d
{
	__global__ void setupEnvironment_d(EModel** qsps, meters* altMin, meters* altMax, double* magnitude, int numRegions)
	{
		ZEROTH_THREAD_ONLY((*qsps) = new QSPS(altMin, altMax, magnitude, numRegions)); //this overloaded constructor is only compiled in the case where __CUDA_ARCH__ is defined
	}

	__global__ void deleteEnvironment_d(EModel** qsps)
	{
		ZEROTH_THREAD_ONLY(delete (*((QSPS**)qsps)));
	}
}

#ifndef __CUDA_ARCH__ //host code
__host__ const vector<meters>& QSPS::altMin() const
{
	return altMin_m;
}

__host__ const vector<meters>& QSPS::altMax() const
{
	return altMax_m;
}

__host__ const vector<double>& QSPS::magnitude() const 
{
	return magnitude_m;
}
#endif


__host__ QSPS::QSPS(meters altMin, meters altMax, Vperm magnitude, int stepUpRegions) : EModel(Type::QSPS)
{
	#ifndef __CUDA_ARCH__ //host code
	altMin_m.push_back(altMin);       //unfortunately this wrapping is necessary
	altMax_m.push_back(altMax);       //as the vectors above also have to be wrapped
	magnitude_m.push_back(magnitude); //in an ifndef/endif block so this will compile

	//step up regions allow the QSPS to gradually step up to full magnitude
	//this avoids a "hard edge" to the QSPS, potentially leading to errors
	if (stepUpRegions != 0)
	{
		constexpr ratio suSize{ 0.05 }; //use step up regions = 5% of QSPS size (arbitrary)

		altMin_m.resize(2 * stepUpRegions + 1); //step up regions on either side of the QSPS, as well as the QSPS itself
		altMax_m.resize(2 * stepUpRegions + 1);
		magnitude_m.resize(2 * stepUpRegions + 1);

		altMin_m.at(stepUpRegions) = altMin; //middle index
		altMax_m.at(stepUpRegions) = altMax;
		magnitude_m.at(stepUpRegions) = magnitude;

		meters size{ altMax - altMin };
		
		for (int iii = 0; iii < stepUpRegions; iii++)
		{//step up regions starting with the bottom of the QSPS ranging to the top
			altMin_m.at(iii) = altMin - (stepUpRegions - iii) * suSize * size;
			altMax_m.at(iii) = altMin - (stepUpRegions - iii - 1) * suSize * size;
			magnitude_m.at(iii) = magnitude * (iii + 1) / (stepUpRegions + 1);
			altMin_m.at(stepUpRegions + 1 + iii) = altMax + iii * suSize * size;
			altMax_m.at(stepUpRegions + 1 + iii) = altMax + (iii + 1) * suSize * size;
			magnitude_m.at(stepUpRegions + 1 + iii) = magnitude * (stepUpRegions - iii) / (stepUpRegions + 1);
		}
	}

	if (useGPU_m) setupEnvironment();
	#endif /* !__CUDA_ARCH__ */
}

__host__ QSPS::QSPS(ifstream& in) : EModel(Type::QSPS)
{
	deserialize(in);
	if (useGPU_m) setupEnvironment();
}

__device__ QSPS::QSPS(meters* altMin, meters* altMax, Vperm* magnitude, int numRegions) : EModel(Type::QSPS),
	altMin_d{ altMin }, altMax_d{ altMax }, magnitude_d{ magnitude }, numRegions_m{ numRegions }
{

}

__host__ __device__ QSPS::~QSPS()
{
	#ifndef __CUDA_ARCH__ //host code
	if (useGPU_m) deleteEnvironment();
	#endif /* !__CUDA_ARCH__ */
}

__host__ void QSPS::setupEnvironment()
{
	#ifndef __CUDA_ARCH__ //host code
	CUDA_API_ERRCHK(hipMalloc((void **)&this_d, sizeof(QSPS*))); //malloc for ptr to ptr to GPU QSPS Obj
	CUDA_API_ERRCHK(hipMalloc((void **)&altMin_d, altMin_m.size() * sizeof(meters))); //array of altitude min bounds
	CUDA_API_ERRCHK(hipMalloc((void **)&altMax_d, altMax_m.size() * sizeof(meters)));
	CUDA_API_ERRCHK(hipMalloc((void **)&magnitude_d, magnitude_m.size() * sizeof(Vperm))); //array of E magnitude between above min/max
	CUDA_API_ERRCHK(hipMemcpy(altMin_d, altMin_m.data(), altMin_m.size() * sizeof(meters), hipMemcpyHostToDevice));
	CUDA_API_ERRCHK(hipMemcpy(altMax_d, altMax_m.data(), altMax_m.size() * sizeof(meters), hipMemcpyHostToDevice));
	CUDA_API_ERRCHK(hipMemcpy(magnitude_d, magnitude_m.data(), magnitude_m.size() * sizeof(meters), hipMemcpyHostToDevice));

	QSPS_d::setupEnvironment_d <<< 1, 1 >>> (this_d, altMin_d, altMax_d, magnitude_d, (int)(magnitude_m.size()));
	CUDA_KERNEL_ERRCHK_WSYNC(); //creates GPU instance of QSPS
	#endif /* !__CUDA_ARCH__ */
}

__host__ void QSPS::deleteEnvironment()
{
	QSPS_d::deleteEnvironment_d <<< 1, 1 >>> (this_d);
	CUDA_KERNEL_ERRCHK_WSYNC();

	CUDA_API_ERRCHK(hipFree(this_d));
	CUDA_API_ERRCHK(hipFree(altMin_d)); //On device
	CUDA_API_ERRCHK(hipFree(altMax_d));
	CUDA_API_ERRCHK(hipFree(magnitude_d));
}

__host__ __device__ Vperm QSPS::getEFieldAtS(const meters s, const seconds t) const
{
	#ifndef __CUDA_ARCH__ //host code
	for (int ind = 0; ind < magnitude_m.size(); ind++)
	{
		if (s >= altMin_m.at(ind) && s <= altMax_m.at(ind))
			return magnitude_m.at(ind);
	}
	#else //device code
	for (int ind = 0; ind < numRegions_m; ind++)
	{
		if (s >= altMin_d[ind] && s <= altMax_d[ind])
			return magnitude_d[ind];
	}
	#endif /* !__CUDA_ARCH__ */

	return 0.0;
}