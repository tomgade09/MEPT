#include "hip/hip_runtime.h"
#include "utils/arrayUtilsGPU.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "hiprand/hiprand_kernel.h"

#include "utils/loopmacros.h"
#include "ErrorHandling/cudaErrorCheck.h"
//#include "ErrorHandling/cudaDeviceMacros.h"

using std::cout;
using std::invalid_argument;

namespace utils
{
	namespace GPU
	{
		__global__ void setup2DArray(double* array1D, double** array2D, int outerDim, int innerDim)
		{//run once on only one thread
			if (blockIdx.x * blockDim.x + threadIdx.x != 0)
				return;

			for (int out = 0; out < outerDim; out++)
				array2D[out] = &array1D[out * innerDim];
		}

		void setup2DArray(double** data1D_d, double*** data2D_d, size_t outerDim, size_t innerDim)
		{
			CUDA_API_ERRCHK(hipMalloc((void**)&(*data1D_d), outerDim * innerDim * sizeof(double*)));
			CUDA_API_ERRCHK(hipMalloc((void**)&(*data2D_d), outerDim * sizeof(double*)));
			
			CUDA_API_ERRCHK(hipMemset(*data1D_d, 0, outerDim * innerDim * sizeof(double)));

			setup2DArray <<< 1, 1 >>> (*data1D_d, *data2D_d, static_cast<int>(outerDim), static_cast<int>(innerDim));
			CUDA_KERNEL_ERRCHK_WSYNC();
		}

		void copy2DArray(vector<vector<double>>& data, double** data1D_d, bool hostToDev)
		{
			size_t frontSize{ data.front().size() };
			for (const auto& elem : data)
				if (elem.size() != frontSize)
					throw invalid_argument("utils::GPU::copy2DArray: inner vectors of argument 'data' (2D double vector) are not equally sized.");

			if (hostToDev)
			{
				LOOP_OVER_1D_ARRAY(data.size(), CUDA_API_ERRCHK(hipMemcpy((*data1D_d) + data.at(0).size() * iii, data.at(iii).data(), data.at(0).size() * sizeof(double), hipMemcpyHostToDevice)));
			}
			else
			{
				LOOP_OVER_1D_ARRAY(data.size(), CUDA_API_ERRCHK(hipMemcpy(data.at(iii).data(), (*data1D_d) + data.at(0).size() * iii, data.at(0).size() * sizeof(double), hipMemcpyDeviceToHost)));
			}
		}

		void free2DArray(double** data1D_d, double*** data2D_d)
		{
			CUDA_API_ERRCHK(hipFree(*data1D_d));
			CUDA_API_ERRCHK(hipFree(*data2D_d));

			*data1D_d = nullptr;
			*data2D_d = nullptr;
		}

		void getGPUMemInfo(size_t* free, size_t* total, int GPUidx)
		{ //use CUDA API to get free and total mem sizes for a specified GPU
			int currDev{ -1 };
			CUDA_API_ERRCHK(hipGetDevice(&currDev));

			if (currDev != GPUidx) CUDA_API_ERRCHK(hipSetDevice(GPUidx));

			CUDA_API_ERRCHK(hipMemGetInfo(free, total));

			if (currDev != GPUidx) CUDA_API_ERRCHK(hipSetDevice(currDev));
		}

		void getCurrGPUMemInfo(size_t* free, size_t* total)
		{ //difference from above is this just checks the current device
			CUDA_API_ERRCHK(hipMemGetInfo(free, total));
		}
	}
}