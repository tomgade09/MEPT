#include "hip/hip_runtime.h"
#include "BField/DipoleB.h"

#include ""
#include "ErrorHandling/cudaErrorCheck.h"
#include "ErrorHandling/cudaDeviceMacros.h"

using std::string;

constexpr double B0{ 3.12e-5 }; //B_0 for Earth dipole B model

//setup CUDA kernels
namespace DipoleB_d
{
	__global__ void setupEnvironmentGPU(BModel** this_d, degrees ILAT, ratio errTol, double ds)
	{
		ZEROTH_THREAD_ONLY((*this_d) = new DipoleB(ILAT, errTol, ds));
	}

	__global__ void deleteEnvironmentGPU(BModel** this_d)
	{
		ZEROTH_THREAD_ONLY(delete ((DipoleB*)(*this_d)));
	}
}

//DipoleB protected member functions
void DipoleB::setupEnvironment()
{// consts: [ ILATDeg, L, L_norm, s_max, ds, lambdaErrorTolerance ]
	CUDA_API_ERRCHK(hipMalloc((void**)&this_d, sizeof(DipoleB*)));
	DipoleB_d::setupEnvironmentGPU <<< 1, 1 >>> (this_d, ILAT_m, lambdaErrorTolerance_m, ds_m);
	CUDA_KERNEL_ERRCHK_WSYNC();
}

void DipoleB::deleteEnvironment()
{
	DipoleB_d::deleteEnvironmentGPU <<< 1, 1 >>> (this_d);
	CUDA_KERNEL_ERRCHK_WSYNC();

	CUDA_API_ERRCHK(hipFree(this_d));
}
//deserialize is in DipoleB.cpp


//DipoleB public member functions
__host__ __device__ DipoleB::DipoleB(degrees ILAT, ratio lambdaErrorTolerance, meters ds, bool useGPU) : BModel(Type::DipoleB),
	ILAT_m{ ILAT }, ds_m{ ds }, lambdaErrorTolerance_m{ lambdaErrorTolerance }, useGPU_m{ useGPU }
{
	L_m = RADIUS_EARTH / (cos(ILAT_m * RADS_PER_DEG) * cos(ILAT_m * RADS_PER_DEG));
	L_norm_m = L_m / RADIUS_EARTH;
	s_max_m = getSAtLambda(ILAT_m);

	#ifndef __CUDA_ARCH__ //host code
	if (useGPU_m) setupEnvironment();
	#endif /* !__CUDA_ARCH__ */
}

__host__ DipoleB::DipoleB(ifstream& in) : BModel(Type::DipoleB)
{
	deserialize(in);
	if (useGPU_m) setupEnvironment();
}

__host__ __device__ DipoleB::~DipoleB()
{
	#ifndef __CUDA_ARCH__ //host code
	if (useGPU_m) deleteEnvironment();
	#endif /* !__CUDA_ARCH__ */
}

__host__ degrees DipoleB::ILAT() const
{
	return ILAT_m;
}

__host__ __device__ meters DipoleB::getSAtLambda(const degrees lambda) const
{
	//double x{ asinh(sqrt(3.0) * sinpi(lambdaDegrees / 180.0)) }; //asinh triggers an odd cuda 8.x bug that is resolved in 9.x+
	double sinh_x{ sqrt(3.0) * sinpi(lambda / 180.0) };
	double x{ log(sinh_x + sqrt(sinh_x * sinh_x + 1)) }; //trig identity for asinh - a bit faster - asinh(x) == ln(x + sqrt(x*x + 1))

	return (0.5 * L_m / sqrt(3.0)) * (x + 0.25 * (exp(2.0*x)-exp(-2.0*x))); /* L */ //0.25 * (exp(2*x)-exp(-2*x)) == sinh(x) * cosh(x) and is faster
}

__host__ __device__ degrees DipoleB::getLambdaAtS(const meters s) const
{
	degrees lambda_tmp{ (-ILAT_m / s_max_m) * s + ILAT_m }; //-ILAT / s_max * s + ILAT
	meters  s_tmp{ s_max_m - getSAtLambda(lambda_tmp) };
	degrees dlambda{ 1.0 };
	bool    over{ 0 };

	while (abs((s_tmp - s) / s) > lambdaErrorTolerance_m)
	{
		while (1)
		{
			over = (s_tmp >= s);
			if (over)
			{
				lambda_tmp += dlambda;
				s_tmp = s_max_m - getSAtLambda(lambda_tmp);
				if (s_tmp < s)
					break;
			}
			else
			{
				lambda_tmp -= dlambda;
				s_tmp = s_max_m - getSAtLambda(lambda_tmp);
				if (s_tmp >= s)
					break;
			}
		}
		if (dlambda < lambdaErrorTolerance_m / 100.0)
			break;
		dlambda /= 5.0; //through trial and error, this reduces the number of calculations usually (compared with 2, 2.5, 3, 4, 10)
	}

	return lambda_tmp;
}

__host__ __device__ tesla DipoleB::getBFieldAtS(const meters s, const seconds simtime) const
{// consts: [ ILATDeg, L, L_norm, s_max, ds, lambdaErrorTolerance ]
	degrees lambda{ getLambdaAtS(s) };
	meters  rnorm{ L_norm_m * cospi(lambda / 180.0) * cospi(lambda / 180.0) };

	return -B0 / (rnorm * rnorm * rnorm) * sqrt(1.0 + 3 * sinpi(lambda / 180.0) * sinpi(lambda / 180.0));
}

__host__ __device__ double DipoleB::getGradBAtS(const meters s, const seconds simtime) const
{
	return (getBFieldAtS(s + ds_m, simtime) - getBFieldAtS(s - ds_m, simtime)) / (2 * ds_m);
}

__host__ __device__ meters DipoleB::getSAtAlt(const meters alt_fromRe) const
{
	degrees lambda{ acos(sqrt((alt_fromRe + RADIUS_EARTH) / L_m)) / RADS_PER_DEG };
	return s_max_m - getSAtLambda(lambda);
}

__host__ ratio  DipoleB::getErrTol() const
{
	return lambdaErrorTolerance_m;
}

__host__ meters DipoleB::getds() const
{
	return ds_m;
}